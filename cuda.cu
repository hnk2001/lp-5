#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matmul(int* A, int* B, int* C, int N) {
    int Row = blockIdx.y*blockDim.y+threadIdx.y;
    int Col = blockIdx.x*blockDim.x+threadIdx.x;
    if (Row < N && Col < N) {
        int Pvalue = 0;
        for (int k = 0; k < N; k++) {
            Pvalue += A[Row*N+k] * B[k*N+Col];
        }
        C[Row*N+Col] = Pvalue;
    }
}

int main() {
    // Example matrices
    const int N = 2;
    int A[N][N] = {{1, 2},
                   {3, 4}};
    int B[N][N] = {{5, 6},
                   {7, 8}};
    int C[N][N];

    // Perform matrix multiplication
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int sum = 0;
            for (int k = 0; k < N; k++) {
                sum += A[i][k] * B[k][j];
            }
            C[i][j] = sum;
        }
    }

    // Print the result
    std::cout << "Result of matrix multiplication:" << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    return 0;
}

// program 2 

#include <iostream>  
#include <hip/hip_runtime.h>

using namespace std;

__global__ void addVectors(int* A, int* B, int* C, int n) 
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; 
    if (i < n) 
    {
        C[i] = A[i] + B[i];
    }
}

int main() 
{
    int n = 1000000;  
    int* A, * B, * C;
    int size = n * sizeof(int);

    // Allocate memory on the host  
    hipHostMalloc(&A, size);  
    hipHostMalloc(&B, size);  
    hipHostMalloc(&C, size);

    // Initialize the vectors
    for (int i = 0; i < n; i++) 
    {
        A[i] = i;
        B[i] = i * 2;
    }
    // Allocate memory on the device  
    int* dev_A, * dev_B, * dev_C;  
    hipMalloc(&dev_A, size);  
    hipMalloc(&dev_B, size);  
    hipMalloc(&dev_C, size);

    // Copy data from host to device
    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);  
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

    // Launch the kernel  
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize;
    addVectors<<<numBlocks, blockSize>>>(dev_A, dev_B, dev_C, n);

    // Copy data from device to host
    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);

    // Print the results
    for (int i = 0; i < 10; i++) 
    {
        cout << C[i] << " ";
    }
    cout << endl;

    // Free memory  
    hipFree(dev_A);  
    hipFree(dev_B);  
    hipFree(dev_C);  
    hipHostFree(A);  
    hipHostFree(B);  
    hipHostFree(C);

    return 0;
}
